#include "hip/hip_runtime.h"
#include <cudaDefs.h>
#include <random>
#include <algorithm>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <benchmark.h>

using gpubenchmark::print_time;

#define COMPUTE_SAFE(x) x

#define M 1000 // Number of individuals
#define n 100    // Dimensionality of world
#define STEP_COUNT 1

typedef struct {
    float* position;
    float* velocity;
    int* target_individuals;
} World;

__device__ World d_world;

__global__ void simulate_step(float* random_vector, int common_target_individual) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < M) {
        // Step 1: Adjust velocities by random vector
        for (int i = 0; i < n; i++) {
            d_world.velocity[tid * n + i] += random_vector[i];
        }

        // Step 2: Move points
        for (int i = 0; i < n; i++) {
            d_world.position[tid * n + i] += d_world.velocity[tid * n + i];
        }

        // Step 3: Move 5% towards pre-selected individuals
        int target = d_world.target_individuals[tid];
        for (int i = 0; i < n; i++) {
            d_world.position[tid * n + i] += 0.05 * (d_world.position[target * n + i] - d_world.position[tid * n + i]);
        }
        
        // Step 4: Move 10% towards common pre-selected individual
        for (int i = 0; i < n; i++) {
            d_world.position[tid * n + i] += 0.1 * (d_world.position[common_target_individual * n + i] - d_world.position[tid * n + i]);
        }
    }
}

int main() {
    // Initialize random vector
    float h_random_vector[n];
    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        h_random_vector[i] = (float)rand() / RAND_MAX;
    }
    float* d_random_vector;
    hipMalloc((void**)&d_random_vector, n * sizeof(float));
    hipMemcpy(d_random_vector, h_random_vector, n * sizeof(float), hipMemcpyHostToDevice);

    // Initialize world
    World h_world;
    h_world.position = (float*)malloc(n * M * sizeof(float));
    h_world.velocity = (float*)malloc(n * M * sizeof(float));
    h_world.target_individuals = (int*)malloc(M * sizeof(int));
    int common_target_individual = rand() % M;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < n; j++) {
            h_world.position[i * n + j] = (float)rand() / RAND_MAX;
            h_world.velocity[i * n + j] = (float)rand() / RAND_MAX;
        }
        h_world.target_individuals[i] = rand() % M;
    }

    /*
    for (int j = 0; j < n; j++)
    {
        printf("%f ", h_random_vector[j]);
    }
    printf("\n");
    printf("\n");

    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", h_world.position[i * n + j]);
            printf("%f ", h_world.velocity[i * n + j]);
        }
        printf("\n");
    }

    printf("\n");
    */

    // Allocate memory on GPU
    World c_world;
    //checkCudaErrors(hipMalloc((void**)&d_world, sizeof(World)));
    checkCudaErrors(hipMalloc(&(c_world.position), n * M * sizeof(float)));
    checkCudaErrors(hipMalloc(&(c_world.velocity), n * M * sizeof(float)));
    checkCudaErrors(hipMalloc(&(c_world.target_individuals), M * sizeof(int)));
    
    checkCudaErrors(hipMemcpy(c_world.position, h_world.position, n * M * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c_world.velocity, h_world.velocity, n * M * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(c_world.target_individuals, h_world.target_individuals, M * sizeof(int), hipMemcpyHostToDevice));
    COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(d_world), &c_world, sizeof(World)));
    //checkCudaErrors(hipMemcpy(&d_world, &h_world, sizeof(World), hipMemcpyHostToDevice));

    // Run simulation
    int num_blocks = (M + 255) / 256;
    int num_threads_per_block = 256;

    auto test1 = [&]()
    {
        simulate_step << <num_blocks, num_threads_per_block >> > (d_random_vector, common_target_individual);
    };

    print_time("simulate_step", test1, 100);
    /*
    for (int step = 0; step < STEP_COUNT; step++) {
        simulate_step << <num_blocks, num_threads_per_block >> > (d_random_vector, common_target_individual);
    }
    */
    hipDeviceSynchronize();

    // Copy data back to host
    //checkCudaErrors(hipMemcpy(&h_world, &c_world, sizeof(World), hipMemcpyDeviceToHost));
    //COMPUTE_SAFE(hipMemcpyToSymbol(HIP_SYMBOL(c_world), &d_world, sizeof(World)));
    checkCudaErrors(hipMemcpy(h_world.position, c_world.position, n * M * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_world.velocity, c_world.velocity, n * M * sizeof(float), hipMemcpyDeviceToHost));

    /*
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", h_world.position[i * n + j]);
            printf("%f ", h_world.velocity[i * n + j]);
        }
        printf("\n");
    }
    */

    // Free memory on GPU
    hipFree(d_world.position);
    hipFree(d_world.velocity);
    hipFree(d_world.target_individuals);
    //hipFree(d_world);

    // Free memory on host
    free(h_world.position);
    free(h_world.velocity);
    free(h_world.target_individuals);

    return 0;
}
